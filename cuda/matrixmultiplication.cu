// Ramon Romero A01700318
// Cuda Lab 2
// 15/11/18

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define NUMBER_OF_THREADS 1024



//iterate over the matrix as it were a unidimensional array
__global__ void matrixmultiplication(int *a, int *b, int *c, int ra, int ca, int rb, int cb){
	int id = threadIdx.x + blockIdx.x* blockDim.x;
	int result = 0;
	while (id < ra * cb){
		int idra = id/cb;
		int idcb = id%cb;

		for (int i = 0; i < ca; i++){
			result+= a[idra * ca + i] * b[cb * i + idcb];
		}

		c[id] = result;
		id += blockDim.x * gridDim.x;
	}
}

int main (){
	printf("\nValidation for a non square matrixmultiplication case (3): \n");


/*
3)

original mats
1.00    2.00
3.00    4.00
5.00    6.00

6.00    7.00    8.00
9.00    0.00    1.00

*/
int rows_a = 3;
int cols_a = 2;
int rows_b = 2;
int cols_b = 3;
	int *dev_a, *dev_b, *dev_c;
	int *a, *b, *c;
	int number_of_threads;

//Validate
	if (cols_a == rows_b){
		a = (int *) malloc(sizeof(int) * rows_a * cols_a);
		b = (int *) malloc(sizeof(int) * rows_b * cols_b);
		c = (int *) malloc(sizeof(int) * rows_a * cols_b);
		hipMalloc((void **)&dev_a, sizeof(int) * rows_a * cols_a);
		hipMalloc((void **)&dev_b, sizeof(int) * rows_b * cols_b);
		hipMalloc((void **)&dev_c, sizeof(int) * rows_a * cols_b);


		a[0]=1.00;
		a[1]=2.00;
		a[2]=3.00;
		a[3]=4.00;
		a[4]=5.00;
		a[5]=6.00;

		b[0]=6.00;
		b[1]=7.00;
		b[2]=8.00;
		b[3]=9.00;
		b[4]=0.00;
		b[5]=1.00;


		printf("\nA: \n\n");
		for (int i = 0; i < rows_a * cols_a; i++){
			if (i%cols_a == 0 && i != 0){
				printf("\n");
			}
			printf("%i ", a[i]);
		}
		printf("\n");

		printf("\nB: \n\n");
		for (int i = 0; i < rows_b * cols_b; i++){
			if (i%cols_b == 0 && i != 0){
				printf("\n");
			}
			printf("%i ", b[i]);
		}
		printf("\n");

		hipMemcpy(dev_a, a, sizeof(int) * rows_a * cols_a, hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, sizeof(int) * rows_b * cols_b, hipMemcpyHostToDevice);



		number_of_threads = (rows_a*cols_b + NUMBER_OF_THREADS - 1) / NUMBER_OF_THREADS;

		dim3 dimGrid(number_of_threads,1,1);
		dim3 dimBlock(NUMBER_OF_THREADS,1,1);



		matrixmultiplication<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, rows_a, cols_a, rows_b, cols_b);


		hipMemcpy(c, dev_c, sizeof(int) * rows_a * cols_b, hipMemcpyDeviceToHost);
		printf("Result Matrix:\n");
		for (int i = 0; i < rows_a * cols_b; i++){
			if (i%cols_b == 0 && i != 0){
				printf("\n");
			}
			printf("%i ", c[i]);
		}
		printf("\n");

		free(a);
		free(b);
		free(c);

		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
	}


	else {
		printf("Cols A must be equal to rows B");
	}
}
