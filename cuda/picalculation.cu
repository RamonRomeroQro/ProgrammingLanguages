// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <https://www.gnu.org/licenses/>.



#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N 1000000
#define width 1.0 / N



__global__ void parallel_pi(double *sum){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int id = index;

    while( id < N){
        double mid = (id + 0.5) * width;
        double height = 4.0 / (1.0 + mid * mid);
        sum[index] = sum[index] + height;
        id = id + blockDim.x * gridDim.x;
    }
}


int main(){
    double area = 0;
    double *p_array;
    double *suma;
    int size = 1000000;
    //allocate array
    suma = (double*) malloc(size * sizeof(double));
    hipMalloc((void**) &p_array, size * sizeof(double));
    parallel_pi<<<int(1000000/1024)+1, 1024>>>(p_array);
    hipMemcpy(suma, p_array, size * sizeof(double), hipMemcpyDeviceToHost);
    //serial suma
    for (int i = 0; i < size; i++) {
        area += suma[i];
    }
    //result
    area = area * width;
    printf("PI= %f \t%i rectangles\n", area, size);
    free(suma);
    hipFree(p_array);

    return 0;
}
