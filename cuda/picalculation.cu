//Jose Ramon Romero Chavez A01700318
//PICUDA 1000000 iterations



#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define N 1000000
#define width 1.0 / N



__global__ void parallel_pi(double *sum){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int id = index;

    while( id < N){
        double mid = (id + 0.5) * width;
        double height = 4.0 / (1.0 + mid * mid);
        sum[index] = sum[index] + height;
        id = id + blockDim.x * gridDim.x;
    }
}


int main(){
    double area = 0;
    double *p_array;
    double *suma;
    int size = 1000000;
    //allocate array
    suma = (double*) malloc(size * sizeof(double));
    hipMalloc((void**) &p_array, size * sizeof(double));
    parallel_pi<<<int(1000000/1024)+1, 1024>>>(p_array);
    hipMemcpy(suma, p_array, size * sizeof(double), hipMemcpyDeviceToHost);
    //serial suma
    for (int i = 0; i < size; i++) {
        area += suma[i];
    }
    //result
    area = area * width;
    printf("PI= %f \t%i rectangles\n", area, size);
    free(suma);
    hipFree(p_array);

    return 0;
}
